#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2020)
 * Projekt c. 1 (cuda)
 * Login: xmarci10
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate velocity and new position for each particle
 * @param p_in  - input particles
 * @param p_out - output particles
 * @param N     - Number of particles
 * @param dt    - Size of the time step
 */
__global__ void calculate_velocity(const t_particles p_in, t_particles p_out, int N, float dt)
{
  extern __shared__ float shared_particles[];

  float *shared_posx = &shared_particles[0];
  float *shared_posy = &shared_particles[blockDim.x];
  float *shared_posz = &shared_particles[2 * blockDim.x];

  float *shared_velx = &shared_particles[3 * blockDim.x];
  float *shared_vely = &shared_particles[4 * blockDim.x];
  float *shared_velz = &shared_particles[5 * blockDim.x];

  float *shared_weight = &shared_particles[6 * blockDim.x];

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  float r, dx, dy, dz;
  float posx, posy, posz;
  float velx, vely, velz;
  float weight;
  float F;

  /**
   * Ressetting the registers for partial results.
   * note:  Using registers reduces the number of accesses to global memory.
   *        Partial results are saved at the end of the calculation.      
   */
  float tmpvelx = 0.0f;
  float tmpvely = 0.0f;
  float tmpvelz = 0.0f;

  /**
   * Loading positions, velocities and weights from the global memory into the registers.
   * note:  Pre-reading data from the global memory, reduces the number of 
   *        memory accesses and thus signigicantly speeds up the calculation.
   */
  posx = (thread_id < N) ? p_in.pos_x[thread_id] : 0.0f;
  posy = (thread_id < N) ? p_in.pos_y[thread_id] : 0.0f;    
  posz = (thread_id < N) ? p_in.pos_z[thread_id] : 0.0f;
  
  velx = (thread_id < N) ? p_in.vel_x[thread_id] : 0.0f;
  vely = (thread_id < N) ? p_in.vel_y[thread_id] : 0.0f;
  velz = (thread_id < N) ? p_in.vel_z[thread_id] : 0.0f;

  weight = (thread_id < N) ? p_in.weight[thread_id] : 0.0f;
  
  // Process the input in the form of "tiles" that are the same size as the blockDim.x.
  for ( struct {int i = 0; int tile = 0;} loop; 
        loop.i < N; 
        loop.i+=blockDim.x, loop.tile++) 
  {
    // Appropriate index into global memory.
    int idx = loop.tile * blockDim.x + threadIdx.x;
    
    /**
     * Loading a single "tile" into shared memory.
     * note:  Pre-reading data from the global memory 2 shared memory, reduces the number of 
     *        memory accesses and thus signigicantly speeds up the calculation.
     */
    shared_posx[threadIdx.x] = (idx < N) ? p_in.pos_x[idx] : 0.0f;
    shared_posy[threadIdx.x] = (idx < N) ? p_in.pos_y[idx] : 0.0f;
    shared_posz[threadIdx.x] = (idx < N) ? p_in.pos_z[idx] : 0.0f;

    shared_velx[threadIdx.x] = (idx < N) ? p_in.vel_x[idx] : 0.0f; 
    shared_vely[threadIdx.x] = (idx < N) ? p_in.vel_y[idx] : 0.0f;
    shared_velz[threadIdx.x] = (idx < N) ? p_in.vel_z[idx] : 0.0f;

    shared_weight[threadIdx.x] =(idx < N) ? p_in.weight[idx] : 0.0f;

    __syncthreads();

    // Process the tile.
    for (int j = 0; j < blockDim.x; j++) {
      /**
       * The calculation of the gravitational force is divided into several
       * several instructions in order to eliminate data dependencies, and thus
       * we have increased the ILP.
       */
      F = -G * dt * shared_weight[j];

      dx = posx - shared_posx[j];
      dy = posy - shared_posy[j];
      dz = posz - shared_posz[j];

      r = sqrt(dx*dx + dy*dy + dz*dz);

      // see previous comment
      F /= (r * r * r + FLT_MIN);

      tmpvelx += (r > COLLISION_DISTANCE) ? F * dx : 0.0f;
      tmpvely += (r > COLLISION_DISTANCE) ? F * dy : 0.0f;
      tmpvelz += (r > COLLISION_DISTANCE) ? F * dz : 0.0f;

      // Add the velocity obtained by the gravitational action of the body 'j'.
      if (r < COLLISION_DISTANCE) {
        /**
         * Reuseage of the registers of distances.
         * note:  The values are calculated only once and then used several times, see below.
         */
        dx = weight - shared_weight[j];
        dy = 2 * shared_weight[j];
        dz = weight + shared_weight[j];

        // Add the velocity obtained by the collision with the body 'j'.
        tmpvelx += (r > 0.0f) ? ((dx * velx + dy * shared_velx[j]) / dz) - velx : 0.0f;
        tmpvely += (r > 0.0f) ? ((dx * vely + dy * shared_vely[j]) / dz) - vely : 0.0f;
        tmpvelz += (r > 0.0f) ? ((dx * velz + dy * shared_velz[j]) / dz) - velz : 0.0f;
      }
    }
    
    __syncthreads();
  }
  /**
   * Update particle
   * note:  Write to global memory only once at the end of the cycle.
   */
  if (thread_id < N) {
    velx += tmpvelx;
    p_out.vel_x[thread_id] = velx;
    p_out.pos_x[thread_id] = velx * dt + posx;

    vely += tmpvely;
    p_out.vel_y[thread_id] = vely;
    p_out.pos_y[thread_id] = vely * dt + posy;

    velz += tmpvelz;
    p_out.vel_z[thread_id] = velz;
    p_out.pos_z[thread_id] = velz * dt + posz;
  }
}// end of calculate_velocity
//-----------------------------------------------------------------------------------------------------------------------

/**
 * Reduction in thread registers. The function uses "shuffle" to exchange data between 
 * threads within the warp (fastest version)
 * 
 * @param val   - each thread in the warp sends its value 'val'
 * @return      - reduced value
 */
__inline__ __device__ float warp_reduce (float val) 
{
  val += __shfl_down_sync(FULL_WARP_MASK, val, 16);
  val += __shfl_down_sync(FULL_WARP_MASK, val, 8);
  val += __shfl_down_sync(FULL_WARP_MASK, val, 4);
  val += __shfl_down_sync(FULL_WARP_MASK, val, 2);
  val += __shfl_down_sync(FULL_WARP_MASK, val, 1);

  return val;
}

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
 __global__ void centerOfMass(const t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
 {
  extern __shared__ float partial_sums[];

  int num_threads = blockDim.x * gridDim.x;  
  int warp_count  = blockDim.x / 32;
  int thread_id   = threadIdx.x;
  int warp_id     = thread_id / 32;
  int lane        = thread_id % 32;

  float *shared_wx = &partial_sums[0];
  float *shared_wy = &partial_sums[1 * warp_count];
  float *shared_wz = &partial_sums[2 * warp_count];
  float *shared_w = &partial_sums[3 * warp_count];

  // Each thread resets its local partial sums
  float wx = 0.0f;
  float wy = 0.0f;
  float wz = 0.0f;
  float w = 0.0f;

  // Reduce multiple elements per thread
  for (int i = thread_id + blockIdx.x * blockDim.x; i < N; i += num_threads) {
    float weight_i = p.weight[i];

    wx += p.pos_x[i] * weight_i;
    wy += p.pos_y[i] * weight_i;
    wz += p.pos_z[i] * weight_i;

    w += weight_i; 
  }

  // Each warp within block performs partial reduction. After this step we get (blockDim.x/32) values. 
  wx = warp_reduce(wx);
  wy = warp_reduce(wy);
  wz = warp_reduce(wz);
  w = warp_reduce(w);

  /**
   * Zero thread within a warp writes the result from the previous step to the shared memory.
   * We write the result on the index of the given warp, by which we ensure the continuos
   * storage of the results.
   */
  if (lane == 0) {
    shared_wx[warp_id] = wx;
    shared_wy[warp_id] = wy;
    shared_wz[warp_id] = wz;
    shared_w[warp_id] = w;
  }

  __syncthreads(); // wait for all partial reductions

/**
 * If the block size is larger than 1024, a reduction in shared memory is used.
 * However, the last 32 values are reduced within one warp (see below).
 *
 * WARNING:   IF U ARE USING BLOCKDIM.X > 1024 CHANGE MAX_BLOCKDIMX VALUE IN nbody.h.
 *            OTHERWISE THE RESULT WILL NOT BE CORRECT.
 */
#if MAX_BLOCKDIMX > 1024 

  for (int stride = warp_count/2; stride > 16; stride >>= 1) {
    if(thread_id < stride) {
      shared_wx[thread_id] += shared_wx[thread_id + stride];
      shared_wy[thread_id] += shared_wy[thread_id + stride];
      shared_wz[thread_id] += shared_wz[thread_id + stride];
      shared_w[thread_id] += shared_w[thread_id + stride]; 
    }
    __syncthreads();
  }

  // First warp loads values from shared memory, so they can be reduces in registers.
  wx = (thread_id < 32) ? shared_wx[thread_id] : 0.0f;
  wy = (thread_id < 32) ? shared_wy[thread_id] : 0.0f;
  wz = (thread_id < 32) ? shared_wz[thread_id] : 0.0f;
  w = (thread_id < 32) ? shared_w[thread_id] : 0.0f;

/**
 * If the block size is less than 1024 then the max number of values after warp
 * reduction will be 32. This means that we can again reduce these values within one 
 * warp. So each thread in the first warp reads the appropriate value from the shared
 * memory into its register if it exists (otherwise zero).
 */
#else

  wx = (thread_id < warp_count) ? shared_wx[thread_id] : 0.0f;
  wy = (thread_id < warp_count) ? shared_wy[thread_id] : 0.0f;
  wz = (thread_id < warp_count) ? shared_wz[thread_id] : 0.0f;
  w = (thread_id < warp_count) ? shared_w[thread_id] : 0.0f;

#endif

  // First warp performs the final reduction.
  if(warp_id == 0) {
    wx = warp_reduce(wx);
    wy = warp_reduce(wy);
    wz = warp_reduce(wz);
    w = warp_reduce(w);
  }

  // Thread 0 writes result into global memory
  if (thread_id == 0) {
    // Write needs to be atomic.
    while(atomicCAS(lock, 0, 1) != 0);
    // critical section
    *comX += wx;
    *comY += wy;
    *comZ += wz;
    *comW += w;
    atomicExch(lock, 0);
  }
 
}// end of centerOfMass
 //----------------------------------------------------------------------------------------------------------------------
 
/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting points and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
