#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Paralelní programování na GPU (PCG 2020)
 * Projekt c. 1 (cuda)
 * Login: xmarci10
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"
#include "h5Helper.h"

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  // Time measurement
  struct timeval t1, t2;

  if (argc != 10)
  {
    printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    exit(1);
  }

  // Number of particles
  const int N           = std::stoi(argv[1]);
  // Length of time step
  const float dt        = std::stof(argv[2]);
  // Number of steps
  const int steps       = std::stoi(argv[3]);
  // Number of thread blocks
  const int thr_blc     = std::stoi(argv[4]);
  // Write frequency
  int writeFreq         = std::stoi(argv[5]);
  // number of reduction threads
  const int red_thr     = std::stoi(argv[6]);
  // Number of reduction threads/blocks
  const int red_thr_blc = std::stoi(argv[7]);

  // Size of the simulation CUDA gird - number of blocks
  const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
  // Size of the reduction CUDA grid - number of blocks
  const size_t reductionGrid  = (red_thr + red_thr_blc - 1) / red_thr_blc;
  // Size of the shared memory used in calculation_velocity kernel  
  const size_t shared_mem_size = thr_blc * 7 * sizeof(float);
  // Size of the shared memory used in centerOfMass kernel  
  const size_t reduction_shared_mem_size = (red_thr_blc/32) * 4 * sizeof(float);

  // Log benchmark setup
  printf("N: %d\n", N);
  printf("dt: %f\n", dt);
  printf("steps: %d\n", steps);
  printf("threads/block: %d\n", thr_blc);
  printf("blocks/grid: %lu\n", simulationGrid);
  printf("reduction threads/block: %d\n", red_thr_blc);
  printf("reduction blocks/grid: %lu\n", reductionGrid);

  const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
  writeFreq = (writeFreq > 0) ?  writeFreq : 0;

  // CPU side memory allocation
  t_particles particles_cpu;
  float4 comOnGPU;

  hipHostAlloc(&particles_cpu.pos_x, N*sizeof(float),hipHostMallocDefault);
  hipHostAlloc(&particles_cpu.pos_y, N*sizeof(float),hipHostMallocDefault);
  hipHostAlloc(&particles_cpu.pos_z, N*sizeof(float),hipHostMallocDefault);
  
  hipHostAlloc(&particles_cpu.vel_x, N*sizeof(float),hipHostMallocDefault);
  hipHostAlloc(&particles_cpu.vel_y, N*sizeof(float),hipHostMallocDefault);
  hipHostAlloc(&particles_cpu.vel_z, N*sizeof(float),hipHostMallocDefault);
  
  hipHostAlloc(&particles_cpu.weight, N*sizeof(float),hipHostMallocDefault);

  /*
   * Caution! Create only after CPU side allocation
   * parameters:
   *                      Stride of two               Offset of the first
   *  Data pointer        consecutive elements        element in floats,
   *                      in floats, not bytes        not bytes
  */
  MemDesc md(
        particles_cpu.pos_x,    1,                          0,              // Postition in X
        particles_cpu.pos_y,    1,                          0,              // Postition in Y
        particles_cpu.pos_z,    1,                          0,              // Postition in Z
        particles_cpu.vel_x,    1,                          0,              // Velocity in X
        particles_cpu.vel_y,    1,                          0,              // Velocity in Y
        particles_cpu.vel_z,    1,                          0,              // Velocity in Z
        particles_cpu.weight,   1,                          0,              // Weight
        N,                                                                  // Number of particles
        recordsNum);                                                        // Number of records in output file

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::cerr<<e.what()<<std::endl;
    return -1;
  }

  // GPU side memory allocation
    // Step 3.*
  float4 *centerOfMassGPU;
  int *lock;

  hipMalloc(&centerOfMassGPU, 4*sizeof(float));
  hipMalloc(&lock, sizeof(int));
    // Step 0-2
  t_particles particles_gpuIn;
  t_particles particles_gpuOut;
  t_particles particles_tmp;

  hipMalloc(&particles_gpuIn.pos_x, N*sizeof(float));
  hipMalloc(&particles_gpuIn.pos_y, N*sizeof(float));
  hipMalloc(&particles_gpuIn.pos_z, N*sizeof(float));
  hipMalloc(&particles_gpuIn.vel_x, N*sizeof(float));
  hipMalloc(&particles_gpuIn.vel_y, N*sizeof(float));
  hipMalloc(&particles_gpuIn.vel_z, N*sizeof(float));
  hipMalloc(&particles_gpuIn.weight, N*sizeof(float));

  hipMalloc(&particles_gpuOut.pos_x, N*sizeof(float));
  hipMalloc(&particles_gpuOut.pos_y, N*sizeof(float));
  hipMalloc(&particles_gpuOut.pos_z, N*sizeof(float));
  hipMalloc(&particles_gpuOut.vel_x, N*sizeof(float));
  hipMalloc(&particles_gpuOut.vel_y, N*sizeof(float));
  hipMalloc(&particles_gpuOut.vel_z, N*sizeof(float));
  hipMalloc(&particles_gpuOut.weight, N*sizeof(float));

  // Transfer data to GPU 
    // Step 3.*
  hipMemset(centerOfMassGPU, 0.0f, 4*sizeof(float));
  hipMemset(lock, 0, sizeof(int));
    // Step 0-2
  hipMemcpy(particles_gpuIn.pos_x, particles_cpu.pos_x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpuIn.pos_y, particles_cpu.pos_y, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpuIn.pos_z, particles_cpu.pos_z, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpuIn.vel_x, particles_cpu.vel_x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpuIn.vel_y, particles_cpu.vel_y, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpuIn.vel_z, particles_cpu.vel_z, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpuIn.weight, particles_cpu.weight, N*sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(particles_gpuOut.weight, particles_cpu.weight, N*sizeof(float), hipMemcpyHostToDevice);

  // Streams and events allocation Step 4
  hipStream_t stream_integrate, stream_com, stream_memcpy;
  hipStreamCreate(&stream_integrate);
  hipStreamCreate(&stream_com);
  hipStreamCreate(&stream_memcpy);

  hipEvent_t particles_finished, com_finished;
  hipEventCreate(&particles_finished);
  hipEventCreate(&com_finished);

  gettimeofday(&t1, 0);

  for(struct {int s = 0; int record_num = 0;} loop; loop.s < steps; loop.s++)
  {
    // Kernel invocation in stream_integrate
    calculate_velocity<<<simulationGrid, thr_blc, shared_mem_size, stream_integrate>>>(particles_gpuIn, particles_gpuOut, N, dt);
    // Inserting an event indicating the completion of the particle position calculation
    hipEventRecord(particles_finished, stream_integrate);

    if (writeFreq > 0 && (loop.s % writeFreq == 0))
    {
      // Kernel invocation in stream_com
      hipMemsetAsync(centerOfMassGPU, 0.0f, 4*sizeof(float), stream_com);
      centerOfMass<<<reductionGrid, red_thr_blc, reduction_shared_mem_size, stream_com>>>(particles_gpuIn, 
        &centerOfMassGPU->x, &centerOfMassGPU->y, &centerOfMassGPU->z, &centerOfMassGPU->w, lock, N);
      // Inserting an event indicating the completion of the center of mass calculation
      hipEventRecord(com_finished, stream_com);

      // Transfer practicles to CPU in stream_memcpy 
      hipMemcpyAsync(particles_cpu.pos_x, particles_gpuIn.pos_x, N*sizeof(float), hipMemcpyDeviceToHost, stream_memcpy);
      hipMemcpyAsync(particles_cpu.pos_y, particles_gpuIn.pos_y, N*sizeof(float), hipMemcpyDeviceToHost, stream_memcpy);
      hipMemcpyAsync(particles_cpu.pos_z, particles_gpuIn.pos_z, N*sizeof(float), hipMemcpyDeviceToHost, stream_memcpy);
      hipMemcpyAsync(particles_cpu.vel_x, particles_gpuIn.vel_x, N*sizeof(float), hipMemcpyDeviceToHost, stream_memcpy);
      hipMemcpyAsync(particles_cpu.vel_y, particles_gpuIn.vel_y, N*sizeof(float), hipMemcpyDeviceToHost, stream_memcpy);
      hipMemcpyAsync(particles_cpu.vel_z, particles_gpuIn.vel_z, N*sizeof(float), hipMemcpyDeviceToHost, stream_memcpy);
      
      // CPU waits until particles data will be available
      hipStreamSynchronize(stream_memcpy);

      // Putting a wait for the com_finished event in the stream_memcpy
      hipStreamWaitEvent(stream_memcpy, com_finished, 0);
      // Transfer com to CPU in stream_memcpy
      hipMemcpyAsync(&comOnGPU.x, centerOfMassGPU, 4*sizeof(float), hipMemcpyDeviceToHost, stream_memcpy);

      // While com is copied from D2H, CPU is writing particles into output file
      h5Helper.writeParticleData(loop.record_num);

      // CPU waits until com data will be available
      // It also ensures that com ends before a new integrate step begins
      hipStreamSynchronize(stream_memcpy);

      // CPU writes the com data into output file
      comOnGPU.x = comOnGPU.x / comOnGPU.w;
      comOnGPU.y = comOnGPU.y / comOnGPU.w;
      comOnGPU.z = comOnGPU.z / comOnGPU.w;
      h5Helper.writeCom(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w, loop.record_num++);
      
    }
    // stream_com needs to wait for an input data
    hipStreamWaitEvent(stream_com, particles_finished, 0);
    // stream_memcpy also needs to wait until the particles computing is finished and after
    // that the data can be copied from D2H
    hipStreamWaitEvent(stream_memcpy, particles_finished, 0);

    // swap pointers
    particles_tmp = particles_gpuOut;
    particles_gpuOut = particles_gpuIn;
    particles_gpuIn = particles_tmp;
  }

  hipDeviceSynchronize();

  hipMemset(centerOfMassGPU, 0.0f, 4*sizeof(float));
  // Kernel invoaction
  centerOfMass<<<reductionGrid, red_thr_blc, reduction_shared_mem_size>>>(particles_gpuIn, 
    &centerOfMassGPU->x, &centerOfMassGPU->y, &centerOfMassGPU->z, &centerOfMassGPU->w, lock, N);

  gettimeofday(&t2, 0);

  // Approximate simulation wall time
  double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
  printf("Time: %f s\n", t);

  // Transfer results back to the CPU 
    // Step 3.*
  hipMemcpy(&comOnGPU.x, centerOfMassGPU, 4*sizeof(float), hipMemcpyDeviceToHost);
    // Step 0-2
  hipMemcpy(particles_cpu.pos_x, particles_gpuIn.pos_x, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.pos_y, particles_gpuIn.pos_y, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.pos_z, particles_gpuIn.pos_z, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.vel_x, particles_gpuIn.vel_x, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.vel_y, particles_gpuIn.vel_y, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.vel_z, particles_gpuIn.vel_z, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.weight, particles_gpuIn.weight, N*sizeof(float), hipMemcpyDeviceToHost);
  
  // CPU completes the calculation of CenterOfMass
  comOnGPU.x = comOnGPU.x / comOnGPU.w;
  comOnGPU.y = comOnGPU.y / comOnGPU.w;
  comOnGPU.z = comOnGPU.z / comOnGPU.w;  
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                         //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float4 comOnCPU = centerOfMassCPU(md);

  std::cout << "Center of mass on CPU:" << std::endl
            << comOnCPU.x <<", "
            << comOnCPU.y <<", "
            << comOnCPU.z <<", "
            << comOnCPU.w
            << std::endl;

  std::cout << "Center of mass on GPU:" << std::endl
            << comOnGPU.x<<", "
            << comOnGPU.y<<", "
            << comOnGPU.z<<", "
            << comOnGPU.w
            << std::endl;

  // Writing final values to the file
  h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
  h5Helper.writeParticleDataFinal();

  // Free CPU memory
  hipHostFree(particles_cpu.pos_x );
  hipHostFree(particles_cpu.pos_y );
  hipHostFree(particles_cpu.pos_z );
  hipHostFree(particles_cpu.vel_x );
  hipHostFree(particles_cpu.vel_y );
  hipHostFree(particles_cpu.vel_z );
  hipHostFree(particles_cpu.weight);
  // Free GPU memory
  hipFree(particles_gpuIn.pos_x);
  hipFree(particles_gpuIn.pos_y);
  hipFree(particles_gpuIn.pos_z);
  hipFree(particles_gpuIn.vel_x);
  hipFree(particles_gpuIn.vel_y);
  hipFree(particles_gpuIn.vel_z);
  hipFree(particles_gpuIn.weight);
  hipFree(particles_gpuOut.pos_x);
  hipFree(particles_gpuOut.pos_y);
  hipFree(particles_gpuOut.pos_z);
  hipFree(particles_gpuOut.vel_x);
  hipFree(particles_gpuOut.vel_y);
  hipFree(particles_gpuOut.vel_z);
  hipFree(particles_gpuOut.weight);
  hipFree(centerOfMassGPU);
  hipFree(lock);

  hipStreamDestroy(stream_integrate);
  hipStreamDestroy(stream_memcpy);
  hipStreamDestroy(stream_com);

  hipEventDestroy(com_finished);
  hipEventDestroy(particles_finished);

  return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
