#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Paralelní programování na GPU (PCG 2020)
 * Projekt c. 1 (cuda)
 * Login: xmarci10
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"
#include "h5Helper.h"

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  // Time measurement
  struct timeval t1, t2;

  if (argc != 10)
  {
    printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    exit(1);
  }

  // Number of particles
  const int N           = std::stoi(argv[1]);
  // Length of time step
  const float dt        = std::stof(argv[2]);
  // Number of steps
  const int steps       = std::stoi(argv[3]);
  // Number of thread blocks
  const int thr_blc     = std::stoi(argv[4]);
  // Write frequency
  int writeFreq         = std::stoi(argv[5]);
  // number of reduction threads
  const int red_thr     = std::stoi(argv[6]);
  // Number of reduction threads/blocks
  const int red_thr_blc = std::stoi(argv[7]);

  // Size of the simulation CUDA gird - number of blocks
  const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
  // Size of the reduction CUDA grid - number of blocks
  const size_t reductionGrid  = (red_thr + red_thr_blc - 1) / red_thr_blc;

  // Log benchmark setup
  printf("N: %d\n", N);
  printf("dt: %f\n", dt);
  printf("steps: %d\n", steps);
  printf("threads/block: %d\n", thr_blc);
  printf("blocks/grid: %lu\n", simulationGrid);
  printf("reduction threads/block: %d\n", red_thr_blc);
  printf("reduction blocks/grid: %lu\n", reductionGrid);

  const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
  writeFreq = (writeFreq > 0) ?  writeFreq : 0;

  // CPU side memory allocation
  t_particles particles_cpu;

  particles_cpu.pos_x = (float*)malloc(N*sizeof(float));
  particles_cpu.pos_y = (float*)malloc(N*sizeof(float));
  particles_cpu.pos_z = (float*)malloc(N*sizeof(float));

  particles_cpu.vel_x = (float*)malloc(N*sizeof(float));
  particles_cpu.vel_y = (float*)malloc(N*sizeof(float));
  particles_cpu.vel_z = (float*)malloc(N*sizeof(float));
  
  particles_cpu.weight = (float*)malloc(N*sizeof(float));

  /*
   * Caution! Create only after CPU side allocation
   * parameters:
   *                      Stride of two               Offset of the first
   *  Data pointer        consecutive elements        element in floats,
   *                      in floats, not bytes        not bytes
  */
  MemDesc md(
        particles_cpu.pos_x,    1,                          0,              // Postition in X
        particles_cpu.pos_y,    1,                          0,              // Postition in Y
        particles_cpu.pos_z,    1,                          0,              // Postition in Z
        particles_cpu.vel_x,    1,                          0,              // Velocity in X
        particles_cpu.vel_y,    1,                          0,              // Velocity in Y
        particles_cpu.vel_z,    1,                          0,              // Velocity in Z
        particles_cpu.weight,   1,                          0,              // Weight
        N,                                                                  // Number of particles
        recordsNum);                                                        // Number of records in output file

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::cerr<<e.what()<<std::endl;
    return -1;
  }


  // GPU side memory allocation
  t_particles particles_gpu;
  
  hipMalloc(&particles_gpu.pos_x, N*sizeof(float));
  hipMalloc(&particles_gpu.pos_y, N*sizeof(float));
  hipMalloc(&particles_gpu.pos_z, N*sizeof(float));
  hipMalloc(&particles_gpu.vel_x, N*sizeof(float));
  hipMalloc(&particles_gpu.vel_y, N*sizeof(float));
  hipMalloc(&particles_gpu.vel_z, N*sizeof(float));
  hipMalloc(&particles_gpu.weight, N*sizeof(float));

  t_velocities tmp_vel;

  hipMalloc(&tmp_vel.x, N*sizeof(float));
  hipMalloc(&tmp_vel.y, N*sizeof(float));
  hipMalloc(&tmp_vel.z, N*sizeof(float));

  // Transfer data to GPU 
  hipMemcpy(particles_gpu.pos_x, particles_cpu.pos_x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpu.pos_y, particles_cpu.pos_y, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpu.pos_z, particles_cpu.pos_z, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpu.vel_x, particles_cpu.vel_x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpu.vel_y, particles_cpu.vel_y, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpu.vel_z, particles_cpu.vel_z, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(particles_gpu.weight, particles_cpu.weight, N*sizeof(float), hipMemcpyHostToDevice);

  hipMemset(tmp_vel.x, 0.0f, N*sizeof(float));
  hipMemset(tmp_vel.y, 0.0f, N*sizeof(float));
  hipMemset(tmp_vel.z, 0.0f, N*sizeof(float));

  gettimeofday(&t1, 0);

  for(int s = 0; s < steps; s++)
  {
    // Kernels invoaction 
    calculate_gravitation_velocity<<<simulationGrid,thr_blc>>>(particles_gpu, tmp_vel, N, dt);
    calculate_collision_velocity<<<simulationGrid,thr_blc>>>(particles_gpu, tmp_vel, N, dt);
    update_particle<<<simulationGrid,thr_blc>>>(particles_gpu, tmp_vel, N, dt);
  }

  hipDeviceSynchronize();

  gettimeofday(&t2, 0);

  // Approximate simulation wall time
  double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
  printf("Time: %f s\n", t);

  float4 comOnGPU;

  // Transfer results back to the CPU 
  hipMemcpy(particles_cpu.pos_x, particles_gpu.pos_x, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.pos_y, particles_gpu.pos_y, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.pos_z, particles_gpu.pos_z, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.vel_x, particles_gpu.vel_x, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.vel_y, particles_gpu.vel_y, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.vel_z, particles_gpu.vel_z, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.weight, particles_gpu.weight, N*sizeof(float), hipMemcpyDeviceToHost);
  
  float4 comOnCPU = centerOfMassCPU(md);

  std::cout << "Center of mass on CPU:" << std::endl
            << comOnCPU.x <<", "
            << comOnCPU.y <<", "
            << comOnCPU.z <<", "
            << comOnCPU.w
            << std::endl;

  std::cout << "Center of mass on GPU:" << std::endl
            << comOnGPU.x<<", "
            << comOnGPU.y<<", "
            << comOnGPU.z<<", "
            << comOnGPU.w
            << std::endl;

  // Writing final values to the file
  h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
  h5Helper.writeParticleDataFinal();

  // Free CPU memory
  free(particles_cpu.pos_x );
  free(particles_cpu.pos_y );
  free(particles_cpu.pos_z );
  free(particles_cpu.vel_x );
  free(particles_cpu.vel_y );
  free(particles_cpu.vel_z );
  free(particles_cpu.weight);
  // Free GPU memory
  hipFree(particles_gpu.pos_x);
  hipFree(particles_gpu.pos_y);
  hipFree(particles_gpu.pos_z);
  hipFree(particles_gpu.vel_x);
  hipFree(particles_gpu.vel_y);
  hipFree(particles_gpu.vel_z);
  hipFree(particles_gpu.weight);
  hipFree(tmp_vel.x);
  hipFree(tmp_vel.y);
  hipFree(tmp_vel.z);

  return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
